#include "hip/hip_runtime.h"
////
////    Matrix Multiplication using CUDA
////    2018202024 �� �� ��
////


#include <hip/hip_runtime.h>
#include <>
//essential cuda headers

#include <iostream>
#include <iomanip>

using namespace std;

__global__ void cudaMatrixMul(int* A, int* B, int* C);

int main()
{
	//width(size) of matrix
	constexpr unsigned int WIDTH = 16;

	//size of Tile
	constexpr unsigned int TILE_WIDTH = 4;

	int matA[WIDTH][WIDTH];
	int matB[WIDTH][WIDTH];
	int matC[WIDTH][WIDTH] = { 0, };

	//initialize matrix
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			matA[y][x] = y * 10 + x;
			matB[y][x] = (y * 10 + x) * 100;
		}
	}
	//end of initializing

	//print matrix
	cout << "Matrix A" << endl;
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			cout << setw(5) << matA[y][x] << ' ';
		}
		cout << endl;
	}

	cout << "Matrix B" << endl;
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			cout << setw(5) << matB[y][x] << ' ';
		}
		cout << endl;
	}

	//allocate space for CUDA (GPU Device)
	int *matA_dev, *matB_dev, *matC_dev;
	hipMalloc(&matA_dev, sizeof(int) * WIDTH * WIDTH);
	hipMalloc(&matB_dev, sizeof(int) * WIDTH * WIDTH);
	hipMalloc(&matC_dev, sizeof(int) * WIDTH * WIDTH);

	//Copy data to device
	hipMemcpy(matA_dev, matA, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(matB_dev, matB, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);

	//launch kernel function
	dim3 tileConf{ WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH };//block configuratio
	dim3 threadConf{ TILE_WIDTH, TILE_WIDTH };//thread configuration inside block
	cudaMatrixMul << < tileConf, threadConf >> > (matA_dev, matB_dev, matC_dev);

	//Synchronize Device
	hipDeviceSynchronize();

	//Gather data from device
	hipMemcpy(matC, matC_dev, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

	//print result
	cout << "\nMultiplication Result" << endl;
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			cout << setw(8) << matC[y][x] << ' ';
		}
		cout << endl;
	}

	//deallocate CUDA(GPU Deivce) space
	hipFree(matA_dev);
	hipFree(matB_dev);
	hipFree(matC_dev);

	return 0;
}

/*
	@name cudaMatrixMul
	@brief matrix parallel multiplication function for square matrix
	@params int *A Matrix A
	@params int *B Matrix B
	@params int *c Result Matrix
*/
__global__ void cudaMatrixMul(int* A, int* B, int* C)
{
	//OK, we need current multiplication target
	//Calculate using CUDA builtin index

	//local thread index
	int x = threadIdx.x;
	int y = threadIdx.y;

	//block index
	int b_x = blockIdx.x;
	int b_y = blockIdx.y;

	//global index
	int gx = b_x * blockDim.x + x;
	int gy = b_y * blockDim.y + y;

	//reset memory
	C[gy * blockDim.y * gridDim.y + gx] = 0;

	//do multiplication
	for (int k = 0; k < blockDim.x * gridDim.x; k++) {//iterate for width of matrix
		int mulR = (gy * blockDim.x * gridDim.x) + k; // = [y][k]
		int mulC = (k * blockDim.x * gridDim.x) + gx; // = [k][x]

		C[gy * blockDim.y * gridDim.y + gx] += A[mulR] * B[mulC];//accumulate result
	}

}