#include <hip/hip_runtime.h>


//headers for CUDA

#include <stdlib.h>
#include <stdio.h>
#define SIZE 10

//CUDA function to run parallely
__global__ void test(int *a, int *b) {
	int i = threadIdx.x;
	b[i] = a[i] + 1;
}

int main() {

	int cnt;
	hipGetDeviceCount(&cnt);
	printf("CUDA Devices found: %d\n", cnt);
	hipDeviceProp_t prop;
	for (int i = 0; i < cnt; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Device #%d name: %s\n", i, prop.name);
	}


	int *a, *b;
	int *d_a, *d_b;

	//allocating space for a and b
	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));

	//allocating space for a and b on GPU
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));

	//setting up values
	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = 0;
	}

	//copy and launch the function
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);//copy to GPU(device) from host
	test <<< 1, SIZE >>> (d_a, d_b); // launch test function
	hipMemcpy(b, d_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);//copy from GPU(device) to host

	for (int i = 0; i < SIZE; i++)
		printf("b[%d] = %d\n", i, b[i]);    // print the results

	free(a);    // free the host memory spaces
	free(b);    // free the host memory spaces

	hipFree(d_a);    // free the device memory spaces 
	hipFree(d_b);    // free the device memory spaces 
	return 0;
}