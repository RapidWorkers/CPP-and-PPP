#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void MatrixMul(float *M, float *N, float *P, int width)
{
	int size = width * width*width;
	float *Md, *Nd, *Pd;

	hipMalloc((void**)Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**)Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);


	hipMalloc((void**)Pd, size);


}